/*
MIT License

Copyright (c) 2025 The-Sycorax (https://github.com/The-Sycorax)

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>

// SHA-256 implementation for CUDA
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))
#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))

typedef unsigned char BYTE;
typedef uint32_t WORD;

typedef struct {
    BYTE data[64];
    WORD datalen;
    unsigned long long bitlen;
    WORD state[8];
} SHA256_CTX;

__constant__ WORD dev_k[64];

static const WORD host_k[64] = {
    0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
    0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
    0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
    0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
    0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
    0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
    0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
    0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

__device__ __forceinline__ void sha256_transform(SHA256_CTX* ctx, const BYTE data[])
{
    WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

    #pragma unroll 16
    for (i = 0, j = 0; i < 16; ++i, j += 4)
        m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);

    #pragma unroll 64
    for (; i < 64; ++i)
        m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

    a = ctx->state[0]; b = ctx->state[1]; c = ctx->state[2]; d = ctx->state[3];
    e = ctx->state[4]; f = ctx->state[5]; g = ctx->state[6]; h = ctx->state[7];

    #pragma unroll 64
    for (i = 0; i < 64; ++i) {
        t1 = h + EP1(e) + CH(e, f, g) + dev_k[i] + m[i];
        t2 = EP0(a) + MAJ(a, b, c);
        h = g; g = f; f = e; e = d + t1;
        d = c; c = b; b = a; a = t1 + t2;
    }

    ctx->state[0] += a; ctx->state[1] += b; ctx->state[2] += c; ctx->state[3] += d;
    ctx->state[4] += e; ctx->state[5] += f; ctx->state[6] += g; ctx->state[7] += h;
}

__device__ __forceinline__ void sha256_init(SHA256_CTX* ctx)
{
    ctx->datalen = 0;
    ctx->bitlen = 0;
    ctx->state[0] = 0x6a09e667;
    ctx->state[1] = 0xbb67ae85;
    ctx->state[2] = 0x3c6ef372;
    ctx->state[3] = 0xa54ff53a;
    ctx->state[4] = 0x510e527f;
    ctx->state[5] = 0x9b05688c;
    ctx->state[6] = 0x1f83d9ab;
    ctx->state[7] = 0x5be0cd19;
}

__device__ __forceinline__ void sha256_update(SHA256_CTX* ctx, const BYTE data[], size_t len)
{
    for (size_t i = 0; i < len; ++i) {
        ctx->data[ctx->datalen] = data[i];
        if (++ctx->datalen == 64) {
            sha256_transform(ctx, ctx->data);
            ctx->bitlen += 512;
            ctx->datalen = 0;
        }
    }
}

__device__ __forceinline__ void sha256_final(SHA256_CTX* ctx, BYTE hash[])
{
    WORD i = ctx->datalen;

    if (ctx->datalen < 56) {
        ctx->data[i++] = 0x80;
        while (i < 56) ctx->data[i++] = 0x00;
    } else {
        ctx->data[i++] = 0x80;
        while (i < 64) ctx->data[i++] = 0x00;
        sha256_transform(ctx, ctx->data);
        for (i = 0; i < 56; ++i) ctx->data[i] = 0x00;
    }

    ctx->bitlen += ctx->datalen * 8;
    ctx->data[63] = ctx->bitlen;
    ctx->data[62] = ctx->bitlen >> 8;
    ctx->data[61] = ctx->bitlen >> 16;
    ctx->data[60] = ctx->bitlen >> 24;
    ctx->data[59] = ctx->bitlen >> 32;
    ctx->data[58] = ctx->bitlen >> 40;
    ctx->data[57] = ctx->bitlen >> 48;
    ctx->data[56] = ctx->bitlen >> 56;
    sha256_transform(ctx, ctx->data);

    for (i = 0; i < 4; ++i) {
        hash[i     ] = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
        hash[i +  4] = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
        hash[i +  8] = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
        hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
    }
}

__device__ __forceinline__ void sha256_to_hex_lc(const unsigned char* data, char* out64)
{
    const char hex[16] = { '0','1','2','3','4','5','6','7','8','9','a','b','c','d','e','f' };
    #pragma unroll
    for (int i = 0; i < 32; ++i) {
        out64[i * 2    ] = hex[(data[i] >> 4) & 0xF];
        out64[i * 2 + 1] = hex[(data[i]     ) & 0xF];
    }
}

__device__ __forceinline__ bool nibble_prefix_match(const char* hh, const unsigned char* chunk, unsigned len)
{
    #pragma unroll
    for (unsigned i = 0; i < len; ++i) { 
        if (hh[i] != (char)chunk[i]) return false; 
    }
    return true;
}

__device__ __forceinline__ bool bytes_contains_lc(const unsigned char* arr, size_t n, unsigned char v)
{
    #pragma unroll
    for (size_t i = 0; i < n; ++i) { 
        if (arr[i] == v) return true; 
    }
    return false;
}

__global__ void miner_kernel(
    const unsigned char* __restrict__ hash_prefix,
    size_t prefix_len,
    const unsigned char* __restrict__ last_chunk,
    unsigned idiff,
    const unsigned char* __restrict__ charset,
    unsigned charset_len,
    unsigned int* __restrict__ result,
    uint32_t start_offset,
    uint32_t global_step,
    uint32_t base_offset,
    uint32_t iters_per_thread
) {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t i = start_offset + tid + base_offset;

    const size_t temp_size = prefix_len + 4;
    unsigned char temp[320];
    unsigned char digest[32];
    char hexhash[64];

    for (size_t k = 0; k < prefix_len; ++k) temp[k] = hash_prefix[k];
    unsigned char* nonce_ptr = temp + prefix_len;

    for (uint32_t it = 0; it < iters_per_thread; ++it) {
        if (*result != 0xFFFFFFFFu) return; // another thread found a result

        // Write nonce in little endian format
        nonce_ptr[0] = (i      ) & 0xFF;
        nonce_ptr[1] = (i >>  8) & 0xFF;
        nonce_ptr[2] = (i >> 16) & 0xFF;
        nonce_ptr[3] = (i >> 24) & 0xFF;

        SHA256_CTX ctx;
        sha256_init(&ctx);
        sha256_update(&ctx, temp, temp_size);
        sha256_final(&ctx, digest);

        sha256_to_hex_lc(digest, hexhash);

        if ((idiff == 0 || nibble_prefix_match(hexhash, last_chunk, idiff)) &&
            (charset_len == 16 || bytes_contains_lc(charset, charset_len, (unsigned char)hexhash[idiff])))
        {
            atomicCAS(result, 0xFFFFFFFFu, i);
            return;
        }

        i += global_step;
    }
}

// C interface for Nim
extern "C" {
    
    hipError_t cuda_miner_init() {
        return hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice);
    }

    hipError_t cuda_mine_nonces(
        const unsigned char* hash_prefix,
        size_t prefix_len,
        const char* last_chunk_str,
        unsigned idiff,
        const char* charset_str,
        unsigned charset_len,
        unsigned int* result,
        uint32_t blocks,
        uint32_t threads,
        uint32_t iters_per_thread,
        uint32_t batch_offset
    ) {
        // Device memory allocation
        unsigned char* d_prefix = nullptr;
        unsigned char* d_last_chunk = nullptr;
        unsigned char* d_charset = nullptr;
        unsigned int* d_result = nullptr;

        hipError_t err;

        // Initialize variables before any goto statements
        unsigned int init_result = 0xFFFFFFFFu;
        uint32_t start_offset = 0;
        uint32_t global_step = blocks * threads;
        uint32_t base_offset = batch_offset * iters_per_thread * global_step;

        // Allocate device memory
        err = hipMalloc(&d_prefix, prefix_len);
        if (err != hipSuccess) return err;

        err = hipMalloc(&d_last_chunk, idiff > 0 ? idiff : 1);
        if (err != hipSuccess) { hipFree(d_prefix); return err; }

        err = hipMalloc(&d_charset, charset_len);
        if (err != hipSuccess) { hipFree(d_prefix); hipFree(d_last_chunk); return err; }

        err = hipMalloc(&d_result, sizeof(unsigned int));
        if (err != hipSuccess) { 
            hipFree(d_prefix); hipFree(d_last_chunk); hipFree(d_charset); 
            return err; 
        }

        // Copy data to device
        err = hipMemcpy(d_prefix, hash_prefix, prefix_len, hipMemcpyHostToDevice);
        if (err != hipSuccess) goto cleanup;

        if (idiff > 0) {
            err = hipMemcpy(d_last_chunk, last_chunk_str, idiff, hipMemcpyHostToDevice);
            if (err != hipSuccess) goto cleanup;
        }

        err = hipMemcpy(d_charset, charset_str, charset_len, hipMemcpyHostToDevice);
        if (err != hipSuccess) goto cleanup;

        err = hipMemcpy(d_result, &init_result, sizeof(unsigned int), hipMemcpyHostToDevice);
        if (err != hipSuccess) goto cleanup;

        // Launch kernel
        miner_kernel<<<blocks, threads>>>(
            d_prefix,
            prefix_len,
            d_last_chunk,
            idiff,
            d_charset,
            charset_len,
            d_result,
            start_offset,
            global_step,
            base_offset,
            iters_per_thread
        );

        err = hipGetLastError();
        if (err != hipSuccess) goto cleanup;

        err = hipDeviceSynchronize();
        if (err != hipSuccess) goto cleanup;

        // Copy result back
        err = hipMemcpy(result, d_result, sizeof(unsigned int), hipMemcpyDeviceToHost);

    cleanup:
        if (d_prefix) hipFree(d_prefix);
        if (d_last_chunk) hipFree(d_last_chunk);
        if (d_charset) hipFree(d_charset);
        if (d_result) hipFree(d_result);

        return err;
    }
}
