#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "sha256.cuh"
#include <string.h>
#include <string>
#include <iostream>
#include <sstream>
#include <iomanip>

void pre_sha256() {
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

__device__ void bytes_slice(const unsigned char* str, unsigned char* buffer, size_t start, size_t end) {
	size_t j = 0;
	for (size_t i = start; i <= end; ++i) {
		buffer[j++] = str[i];
	}
	buffer[j] = 0;
}

__device__ bool my_strcmp(const char* str_a, const unsigned char* str_b, unsigned len) {
	for (int i = 0; i < len; i++) {
		if (str_a[i] != str_b[i])
			return false;
	}
	return true;
}

__device__ void sha256_to_hex(const unsigned char* data, char pout[64])
{

	const char hex[16] = { '0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'A', 'B', 'C', 'D', 'E', 'F' };
	int i = 0;
	for (; i < 32; ++i) {
		pout[i * 2] = hex[(data[i] >> 4) & 0xF];
		pout[i * 2 + 1] = hex[(data[i]) & 0xF];
	}
}

__device__ bool bytes_contains(const unsigned char* str_1, size_t str_1_len, unsigned char str_2) {
	for (int i = 0; i < str_1_len; i++) {
		if (str_1[i] == str_2)
			return true;
	}
	return false;
}

__device__ void compute_hash(void* input, size_t input_size, unsigned char* out) {
	SHA256_CTX ctx;
	sha256_init(&ctx);
	sha256_update(&ctx, (unsigned char*)input, input_size);
	sha256_final(&ctx, out);
}

__device__ void lprint(const char* data, size_t length) {
	for (int x = 0; x < length; x++)
		printf("%c", data[x]);
	printf("\n");
}

__global__ void miner_thread(const unsigned char* hash_prefix, const unsigned char* last_block_chunk, size_t hash_prefix_length, size_t difficulty, const unsigned char* charset, size_t charset_len, int *stop, int step) {
	
	// Starting point of i.
	uint64_t start = blockIdx.x * blockDim.x + threadIdx.x;

	// Distribution algorithm (5 threads example):
	// 0 1 2 3 4 (starting points)
	// 5 6 7 8 9 (adding the step which is the number of threads)
	// 10 11 12 13 14 ...

	// Allocating local variables.
	size_t temp_size = hash_prefix_length + 4;
	uint32_t i = start;
	unsigned char temp[300];
	
	// Copying hash prefix to temp
	memcpy(temp, hash_prefix, hash_prefix_length);
	unsigned char* nonce_pointer = temp + hash_prefix_length;

	unsigned char out[32];
	char hash_hex[64];


	while (*stop == 0) {
		// Adding hash random to temp
		memcpy(nonce_pointer, &i, 4);

		// Computing hash
		compute_hash(temp, temp_size, out);

		// Turning it to hex
		sha256_to_hex(out, hash_hex);

		// Checking if it's valid
		if (my_strcmp(hash_hex, last_block_chunk, difficulty) && bytes_contains(charset, charset_len, hash_hex[difficulty])) {
			// If it's valid stop all threads and print the random.
			printf("%d\n", i);
			*stop = -1;
			break;
		} else if (i == 4294967295) {
			// If it reaches the uint32 maximum print 0 and stop all threads.
			*stop = -1;
			printf("0\n");
			break;
		}
		// Add the step to i (blocks * threads)
		i += step;
	}
}

int char2int(char input) {
	if (input >= '0' && input <= '9')
		return input - '0';
	if (input >= 'A' && input <= 'F')
		return input - 'A' + 10;
	if (input >= 'a' && input <= 'f')
		return input - 'a' + 10;
	throw std::invalid_argument("Invalid input string");
}

void hex2bin(const char* src, char* target) {
	while (*src && src[1])
	{
		*(target++) = char2int(*src) * 16 + char2int(src[1]);
		src += 2;
	}
}

void hex_print(const unsigned char* data, size_t length) {
	for (int x = 0; x < length; x++)
		printf("%02X", data[x]);
	printf("\n");
}

int main(int argc, char** argv) {

	if (argc < 5)
		return -1;

	// Settings
	int blocks = 50;
	int threads = 512;

	// Console arguments (last_block_chunk charset hex_hash_prefix difficulty)
	int difficulty = std::stoi(argv[4]);
	std::string _last_block_chunk(argv[1]);
	std::string _charset(argv[2]);
	std::string hex_hash_prefix(argv[3]);
	size_t hash_prefix_length = (size_t) (hex_hash_prefix.length() / 2);

	// Hex hash prefix to bytes hash prefix
	unsigned char* hash_prefix;
	char* temp_hash_prefix;
	temp_hash_prefix = static_cast<char*>(malloc(hash_prefix_length));
	hipMallocManaged(&hash_prefix, hash_prefix_length);
	hex2bin(hex_hash_prefix.c_str(), temp_hash_prefix);
	hipMemcpy(hash_prefix, (unsigned char *) temp_hash_prefix, hash_prefix_length, hipMemcpyHostToDevice);
	free(temp_hash_prefix);

	// Allocating global memory variables
	unsigned char* last_block_chunk;
	unsigned char* charset;
	int* stop;
	size_t charset_length = _charset.length();

	pre_sha256();
	hipMallocManaged(&stop, sizeof(int));
	hipMallocManaged(&charset, charset_length);
	hipMallocManaged(&last_block_chunk, difficulty);
	
	// Copying to global memory variables
	hipMemcpy(charset, _charset.c_str(), charset_length, hipMemcpyHostToDevice);
	hipMemcpy(last_block_chunk, _last_block_chunk.c_str(), difficulty, hipMemcpyHostToDevice);
	*stop = 0;

	int step = blocks * threads;
	// GPU starts

	// Starting threads.
	miner_thread <<<blocks, threads>>> (hash_prefix, last_block_chunk, hash_prefix_length, (size_t)difficulty, charset, charset_length, stop, step);

	// Waiting for completion, and verifying if there was any error.
	checkCudaErrors(hipDeviceSynchronize());

	//GPU ends


	// Freeing global variables...
	hipFree(hash_prefix);
	hipFree(stop);
	hipFree(charset);
	hipFree(last_block_chunk);

	return 0;
}
